#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

//************variables globales***************
int msk=3, dimx=1040, dimy=1388, tam_imag=1388*1040;

// [i][j] = i*dimy+j

//************** Kernel CUDA *********************
__global__ void EDF(int *R_d,int *G_d,int *B_d,int *Rf_d,int *Gf_d,int *Bf_d,int *topof_d,float *max_d){

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int id = idx + idy*blockDim.x*gridDim.x;

	int dimx=1040, dimy=1388, tam_imag=1388*1040, msk=3, M_d[9], k;
	float X=0.f,Xprom=0.f,Y=0.f, var=0.f;

	if(id<tam_imag){
		M_d[0]=((idx<1 || idy<1) ? 0:G_d[(idx-1)+(idy-1)]*blockDim.x*gridDim.x);
		M_d[1]=((idx<1) ? 0:G_d[(idx-1)+(idy)]*blockDim.x*gridDim.x);
		M_d[2]=((idx<1 || idy>dimy-2) ? 0:G_d[(idx-1)+(idy+1)*blockDim.x*gridDim.x]);
		M_d[3]=((idy<1) ? 0:G_d[(idx)+(idy-1)*blockDim.x*gridDim.x]);
		M_d[4]=G_d[(idx)+(idy)*blockDim.x*gridDim.x];
		M_d[5]=((idy>dimy-2) ? 0:G_d[(idx)+(idy+1)*blockDim.x*gridDim.x]);
		M_d[6]=((idx>dimx-2 || idy<1) ? 0:G_d[(idx+1)+(idy-1)*blockDim.x*gridDim.x]);
		M_d[7]=((idx>dimx-2) ? 0:G_d[(idx+1)+(idy)*blockDim.x*gridDim.x]);
		M_d[8]=((idx>dimx-2 || idy>dimy-1) ? 0:G_d[(idx+1)+(idy+1)*blockDim.x*gridDim.x]);

		for(k=0;k<msk*msk;k++)
			X+=M_d[k];
				
		Xprom=((float)X)/(msk*msk);

		for(k=0;k<msk*msk;k++)
			Y+=(Xprom-M_d[k])*(Xprom-M_d[k]);

		var=Y/(msk*msk);

		syncthreads(); //Barrera

		if(var>max_d[id]){
			topof_d[id]=id;
			Rf_d[id]=R_d[id];
			Gf_d[id]=G_d[id];
			Bf_d[id]=B_d[id];
			max_d[id]=var;
		}
	}
}



//*****************Funcion main**********************
int main(int argc,char* argv[]){

	//***************Declaracion de variables**************
	int i,j,m,cont,tam_B, init,fin;
	//init=atoi(argv[1]);
	//fin=atoi(argv[2]);

	init=1;
	fin=328;

	FILE *matrizR, *matrizG, *matrizB;
	int d;
	float t;
	clock_t tinicio, t_GPU;
	tinicio=clock();

	/* //Tipo MATRIZ

	int **topof_h, **R_h, **G_h, **B_h, **Rf_h, **Gf_h, **Bf_h;
	float **max_h, *var_h;

	int **topof_d, **R_d, **G_d, **B_d, **Rf_d, **Gf_d, **Bf_d;
	float **max_d, *var_d;
	*/

	int *topof_h, *R_h, *G_h, *B_h, *Rf_h, *Gf_h, *Bf_h;
	float *max_h;
	//, *var_h;

	int *topof_d, *R_d, *G_d, *B_d, *Rf_d, *Gf_d, *Bf_d;
	float *max_d;
	//, *var_d;
	
	//************Inicializacion de variables en el host y en el device *************** 
	
	/* // Declaracion tipo MATRIZ
	max_h=(float **)malloc(sizeof(float)*dimx);
	topof_h=(int **)malloc(sizeof(int)*dimx);
	R_h=(int **)malloc(sizeof(int)*dimx);
	G_h=(int **)malloc(sizeof(int)*dimx);
	B_h=(int **)malloc(sizeof(int)*dimx);
	Rf_h=(int **)malloc(sizeof(int)*dimx);
	Gf_h=(int **)malloc(sizeof(int)*dimx);
	Bf_h=(int **)malloc(sizeof(int)*dimx);
	for(i=0;i<dimx;i++){
		max_h[i]=(float*)malloc(sizeof(float)*dimy);
		topof_h[i]=(int*)malloc(sizeof(int)*dimy);
		R_h[i]=(int*)malloc(sizeof(int)*dimy);
		G_h[i]=(int*)malloc(sizeof(int)*dimy);
		B_h[i]=(int*)malloc(sizeof(int)*dimy);
		Rf_h[i]=(int*)malloc(sizeof(int)*dimy);
		Gf_h[i]=(int*)malloc(sizeof(int)*dimy);
		Bf_h[i]=(int*)malloc(sizeof(int)*dimy);
	}
	var_h=(float *)malloc(sizeof(float)*tam_imag);
	*/

	R_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&R_d, tam_imag*sizeof(int));
	G_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&G_d, tam_imag*sizeof(int));
	B_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&B_d, tam_imag*sizeof(int));
	Rf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Rf_d, tam_imag*sizeof(int));
	Gf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Gf_d, tam_imag*sizeof(int));
	Bf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Bf_d, tam_imag*sizeof(int));
	topof_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&topof_d, tam_imag*sizeof(int));

	max_h=(float *)malloc(sizeof(float)*tam_imag);
	hipMalloc((void**)&max_d, tam_imag*sizeof(float));
	//var_h=(float *)malloc(sizeof(float)*tam_imag);
	//hipMalloc((void**)&var_d, tam_imag*sizeof(float));

	//Ejemplos +++++++++++++++
	//memset(str,'$',7);
	//hipMemset((void *) d_array, 0, ARRAY_BYTES);
	hipMemset((void *) max_h, 0, sizeof(float)*tam_imag);

	
	//*************** For cálculo EDF ****************
	for(d=init;d<=fin;d++){

		printf("d=%d \n", d);
		
		//*****************Lecura de matrices RGB en el host****************

		char rutaR[]="";
		sprintf(rutaR, "%s%d%s","RGB/",d,"/R"); 
		matrizR=fopen(rutaR,"r+");

		char rutaG[]="";
		sprintf(rutaG, "%s%d%s","RGB/",d,"/G"); 
		matrizG=fopen(rutaG,"r+");

		char rutaB[]="";
		sprintf(rutaB, "%s%d%s","RGB/",d,"/B"); 
		matrizB=fopen(rutaB,"r+");

		for(i=0;i<dimx;i++){
			for(j=0;j<dimy;j++){
				fscanf(matrizR, "%d", &R_h[i*dimy+j]);
				fscanf(matrizG, "%d", &G_h[i*dimy+j]); 
				fscanf(matrizB, "%d", &B_h[i*dimy+j]); 
			}
		}
		fclose(matrizR);
		fclose(matrizG);
		fclose(matrizB);

		//***************** Kernel EDF *******************

		hipMemcpy(R_d,R_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(G_d,G_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(B_d,B_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);

		hipMemcpy(Rf_d,Rf_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(Gf_d,Gf_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(Bf_d,Bf_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);

		hipMemcpy(topof_d,topof_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(max_d,max_h,sizeof(float)*tam_imag,hipMemcpyHostToDevice);

		dim3 Grid(347,20);
		dim3 Block(13,16);

		//EDF<<<Grid,Block>>>(R_d,G_d,B_d,Rf_d,Gf_d,Bf_d,topof_d,max_d,var_d);
		EDF<<<Grid,Block>>>(R_d,G_d,B_d,Rf_d,Gf_d,Bf_d,topof_d,max_d);

		hipMemcpy(Rf_h,Rf_d,sizeof(int)*tam_imag,hipMemcpyDeviceToHost);
		hipMemcpy(Gf_h,Gf_d,sizeof(int)*tam_imag,hipMemcpyDeviceToHost);
		hipMemcpy(Bf_h,Bf_d,sizeof(int)*tam_imag,hipMemcpyDeviceToHost);

		hipMemcpy(topof_h,topof_d,sizeof(int)*tam_imag,hipMemcpyDeviceToHost);
		hipMemcpy(max_h,max_d,sizeof(float)*tam_imag,hipMemcpyDeviceToHost);



	} //Finaliza For cálculo EDF

	//****************Almacenamiento matrices**************

	FILE *archTopo, *archR, *archG, *archB;
	archTopo=fopen("Resultados/topos10","w+");
	archR=fopen("Resultados/R10","w+");
	archG=fopen("Resultados/G10","w+");
	archB=fopen("Resultados/B10","w+");
	for(i=0;i<dimx;i++){
		for(j=0;j<dimy;j++){
			fprintf(archTopo,"%d ",topof_h[i*dimy+j]);
			fprintf(archR,"%d ",Rf_h[i*dimy+j]);
			fprintf(archG,"%d ",Gf_h[i*dimy+j]);
			fprintf(archB,"%d ",Bf_h[i*dimy+j]);
			/*if(j%2559==0 && j!=0){
				fprintf(topo,"\n");
			}*/
		}
		fprintf(archTopo,"\n");
		fprintf(archR,"\n");
		fprintf(archG,"\n");
		fprintf(archB,"\n");
	}
	fclose(archTopo);
	fclose(archR);
	fclose(archG);
	fclose(archB);

	//free(var_h);
	free(max_h);
	free(topof_h);
	free(R_h);
	free(G_h);
	free(B_h);
	free(Rf_h);
	free(Gf_h);
	free(Bf_h);

	//cudafree(var_d);
	hipFree(max_d);
	hipFree(topof_d);
	hipFree(R_d);
	hipFree(G_d);
	hipFree(B_d);
	hipFree(Rf_d);
	hipFree(Gf_d);
	hipFree(Bf_d);

	
	t_GPU=clock();
	t = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("\ntiempo de procesamiento de varianzas: %6.3fs\n",t);

	getchar ();
	return 0;

}//FIN funcion main()
