#include "hip/hip_runtime.h"
/*
The code generates a 3D image of a stack of images.

For each image (matrix) calculate the variance at all points, and then create a topography matrix (relief matrix) with
the position (number in the stack) of the image that had the largest variance in a pixel. The same with the color of the
image (RGB matrices).
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

//************Global variables***************

struct point
{
	int x;
	int y;
};
#define IJ_TO_ID(i,j) (((i)*dimy)+(j))

//************** Kernel CUDA *********************
__global__  void EDF(int *R_d, int *G_d, int *B_d, int *Rf_d, int *Gf_d, int *Bf_d, int *topof_d, long double *max_d, int d) {

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int id = idx + idy*blockDim.x*gridDim.x;

	//int id = idy + idx*blockDim.y*gridDim.y;

	int dimx = 1040, dimy = 1388, tam_imag = 1388 * 1040, msk = 3, M_d[9], k;
	long double X = 0.f, Xprom = 0.f, Y = 0.f, var = 0.f;
	//Rf_d[id] = id;
	//int img_x = (id) % dimx;
	//int img_y = (id) / dimx;

	int img_x = id / dimy;
	int img_y = id % dimy;

	//int i = 0;
	//unsigned long long int id2;

	M_d[0] = ((img_x < 1 || img_y < 1) ? 0 : G_d[IJ_TO_ID(img_x - 1, img_y - 1)]);
	M_d[1] = ((img_x < 1) ? 0 : G_d[IJ_TO_ID(img_x - 1, img_y)]);
	M_d[2] = ((img_x<1 || img_y>dimy - 2) ? 0 : G_d[IJ_TO_ID(img_x - 1, img_y + 1)]);
	M_d[3] = ((img_x < 1) ? 0 : G_d[IJ_TO_ID(img_x, img_y - 1)]);
	M_d[4] = G_d[IJ_TO_ID(img_x, img_y)];
	M_d[5] = ((img_y > dimy - 2) ? 0 : G_d[IJ_TO_ID(img_x, img_y + 1)]);
	M_d[6] = ((img_x > dimx - 2 || img_y < 1) ? 0 : G_d[IJ_TO_ID(img_x + 1, img_y - 1)]);
	M_d[7] = ((img_x > dimx - 2) ? 0 : G_d[IJ_TO_ID(img_x + 1, img_y)]);
	M_d[8] = ((img_x > dimx - 2 || img_y > dimy - 2) ? 0 : G_d[IJ_TO_ID(img_x + 1, img_y + 1)]);


	for (k = 0;k < msk*msk;k++)
		X += M_d[k];

	Xprom = ((long double)X) / (msk*msk);

	for (k = 0;k < msk*msk;k++)
		Y += (Xprom - M_d[k])*(Xprom - M_d[k]);
	
	var = ((long double)Y) / (msk*msk);
	//syncthreads();
	__syncthreads();
	//hosam

	if (var > max_d[id]) {
		topof_d[id] = d;
		Rf_d[id] = R_d[id];
		Gf_d[id] = G_d[id];
		Bf_d[id] = B_d[id];
		max_d[id] = var;
	}
}


long msk = 3, dimx = 1040, dimy = 1388, tam_imag = 1388 * 1040;
//*****************Main function**********************
int main(int argc, char* argv[]) {

	//***************Variables**************
	int i, j, m, cont, tam_B, init, fin;
	hipError_t cudaStatus;
	FILE *matrizR, *matrizG, *matrizB;
	int d;
	float t;
	clock_t tinicio, t_GPU;
	tinicio = clock();

	int *topof_h, *R_h, *G_h, *B_h, *Rf_h, *Gf_h, *Bf_h;
	long double *max_h;

	int *topof_d, *R_d, *G_d, *B_d, *Rf_d, *Gf_d, *Bf_d;
	long double *max_d;

	//************ Malloc in host and device *************** 
	R_h = (int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&R_d, tam_imag * sizeof(int));
	G_h = (int *)malloc(sizeof(int)*tam_imag);
	cudaStatus = hipMalloc((void**)&G_d, tam_imag * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for G_d Line=%d!\n", __LINE__);
		exit(0);
	}
	B_h = (int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&B_d, tam_imag * sizeof(int));

	Rf_h = (int *)malloc(sizeof(int)*tam_imag);
	memset((void*)Rf_h, 0, sizeof(int)*tam_imag);
	cudaStatus = hipMalloc((void**)&Rf_d, tam_imag * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for Rf_d!\n");
		exit(0);
	}
	Gf_h = (int *)malloc(sizeof(int)*tam_imag);
	memset((void*)Gf_h, 0, sizeof(int)*tam_imag);
	cudaStatus = hipMalloc((void**)&Gf_d, tam_imag * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
	}
	Bf_h = (int *)malloc(sizeof(int)*tam_imag);
	memset((void*)Bf_h, 0, sizeof(int)*tam_imag);
	cudaStatus = hipMalloc((void**)&Bf_d, tam_imag * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
	}
	topof_h = (int *)malloc(sizeof(int)*tam_imag);
	memset((void *)topof_h, 0, sizeof(int)*tam_imag);
	cudaStatus = hipMalloc((void**)&topof_d, tam_imag * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
	}

	//hipMemset((void *)topof_d, 0, tam_imag * sizeof(int)); //hosam

	max_h = (long double *)malloc(sizeof(long double)*tam_imag);
	memset((void*)max_h, 0, sizeof(long double)*tam_imag);
	cudaStatus = hipMalloc((void**)&max_d, tam_imag * sizeof(long double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
	}
	//hipMemset(max_d, 0, sizeof(float)*tam_imag);

	//hipMemset((void *)max_h, 0, sizeof(float)*tam_imag);

	//init=atoi(argv[1]);
	//fin=atoi(argv[2]);
	init = 1;
	fin = 20;

	//*************** Principal FOR ****************
	for (d = init;d <= fin;d++)
	{

		printf("d=%d \n", d);
		//*****************Read RGB files****************
		char rutaR[1024];
		sprintf(rutaR, "%s%d%s", "RGB/", d, "/R");
		matrizR = fopen(rutaR, "r+");
		char rutaG[1024];
		sprintf(rutaG, "%s%d%s", "RGB/", d, "/G");
		matrizG = fopen(rutaG, "r+");
		if (!matrizG)
		{
			printf("Error open file \n");
			exit(0);
		}

		char rutaB[1024];
		sprintf(rutaB, "%s%d%s", "RGB/", d, "/B");
		matrizB = fopen(rutaB, "r+");
		memset((void*)R_h, 0, sizeof(int)*tam_imag);
		memset((void*)G_h, 0, sizeof(int)*tam_imag);
		memset((void*)B_h, 0, sizeof(int)*tam_imag);
		for (i = 0;i < dimx;i++)
		{
			for (j = 0;j < dimy;j++)
			{
				fscanf(matrizR, "%d", &R_h[i*dimy + j]);
				fscanf(matrizG, "%d", &G_h[i*dimy + j]);
				fscanf(matrizB, "%d", &B_h[i*dimy + j]);
			}
		}
		fclose(matrizR);
		fclose(matrizG);
		fclose(matrizB);

		//***************** Kernel EDF *******************

		cudaStatus = hipMemcpy(R_d, R_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed for %d!\n", __LINE__);
			exit(0);
		}

		cudaStatus = hipMemcpy(G_d, G_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus = hipMemcpy(B_d, B_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		cudaStatus = hipMemcpy(Rf_d, Rf_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus = hipMemcpy(Gf_d, Gf_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus = hipMemcpy(Bf_d, Bf_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		cudaStatus = hipMemcpy(topof_d, topof_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus = hipMemcpy(max_d, max_h, sizeof(long double)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		dim3 Grid(347, 20);
		dim3 Block(13, 16);

		EDF << <Grid, Block >> > (R_d, G_d, B_d, Rf_d, Gf_d, Bf_d, topof_d, max_d, d);

		//printf("\n\n FINISH \n\n");
		//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ The code stops here
		cudaStatus = hipMemcpy(Rf_h, Rf_d, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus = hipMemcpy(Gf_h, Gf_d, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus = hipMemcpy(Bf_h, Bf_d, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		cudaStatus = hipMemcpy(topof_h, topof_d, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus = hipMemcpy(max_h, max_d, sizeof(long double)*tam_imag, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

	} //End for

	  //****************Save results**************
	FILE *archTopo, *archR, *archG, *archB;
	archTopo = fopen("Resultados/topo-f2.txt", "w+");
	archR = fopen("Resultados/R-f2.txt", "w+");
	archG = fopen("Resultados/G-f2.txt", "w+");
	archB = fopen("Resultados/B-f2.txt", "w+");
	for (i = 0;i < dimx;i++) {
		for (j = 0;j < dimy;j++) {
			fprintf(archTopo, "%d ", topof_h[i*dimy + j]);
			fprintf(archR, "%d ", Rf_h[i*dimy + j]);
			fprintf(archG, "%d ", Gf_h[i*dimy + j]);
			fprintf(archB, "%d ", Bf_h[i*dimy + j]);
		}
		fprintf(archTopo, "\n");
		fprintf(archR, "\n");
		fprintf(archG, "\n");
		fprintf(archB, "\n");
	}
	/*for (i = 0;i < tam_imag;i++) {

			fprintf(archTopo, "%d ", topof_h[i]);
			fprintf(archR, "%d ", Rf_h[i]);
			fprintf(archG, "%d ", Gf_h[i]);
			fprintf(archB, "%d ", Bf_h[i]);

	}*/

	fclose(archTopo);
	fclose(archR);
	fclose(archG);
	fclose(archB);

	free(max_h);
	free(topof_h);
	free(R_h);
	free(G_h);
	free(B_h);
	free(Rf_h);
	free(Gf_h);
	free(Bf_h);

	hipFree(max_d);
	hipFree(topof_d);
	hipFree(R_d);
	hipFree(G_d);
	hipFree(B_d);
	hipFree(Rf_d);
	hipFree(Gf_d);
	hipFree(Bf_d);


	t_GPU = clock();
	t = ((float)t_GPU - (float)tinicio) / CLOCKS_PER_SEC;
	printf("\ntiempo de procesamiento de varianzas: %6.3fs\n", t);

	//getchar ();
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;

}//END Main function
