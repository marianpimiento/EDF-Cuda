#include "hip/hip_runtime.h"
//*************incluci�n de librerias***************

#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>

//************variables globales***************

int N=93, dimx=1920, dimy=2560, tam_imag=1920*2560;

//**********KERNEL**************
__global__ void kernel (float *max, float *var, int *top, int k){
	int idx=threadIdx.x + blockIdx.x*blockDim.x;
	int tam_imag=1920*2560;

	if(idx<tam_imag){
		if(var[idx]>max[idx]){
			top[idx]=k;
			max[idx]=var[idx];
		}
	}
}

float *leerMatrizVarianza(int d);

//*****************funci�n main**********************

int main(int argc,char* argv[]){

	//***************declaración de variables**************

	int i,k,temp;
	int *top_d; int top_h[dimx*dimy];
	hipMalloc((void **)&top_d,sizeof(int)*dimx*dimy);

	float *max_d, *var_d;
	float *max_h, *var_h;

	var_h=(float *)malloc(sizeof(float)*dimx*dimy)
	max_h=(float *)malloc(sizeof(float)*dimx*dimy);
	hipMalloc((void **)&max_d,sizeof(float)*dimx*dimy);
	hipMalloc((void **)&var_d,sizeof(float)*dimx*dimy);

	float t;
	clock_t tinicio, t_GPU;
	tinicio=clock();

	//***************c�lculo de la mayor varianza************

	temp=1;
	max_h=leerMatrizVarianza(temp);
	for(i=0;i<dimx*dimy;i++)
		top_h[i]=temp;
		for(k=2;k<=N;k++){
			printf("k=%d\n", k);
			var_h=leerMatrizVarianza(k);
			hipMemcpy(max_d,max_h,sizeof(float)*dimx*dimy,hipMemcpyHostToDevice);
			hipMemcpy(var_d,var_h,sizeof(float)*dimx*dimy,hipMemcpyHostToDevice);
			hipMemcpy(top_d,top_h,sizeof(int)*dimx*dimy,hipMemcpyHostToDevice);

			kernel<<<12288,400>>>(max_d,var_d,top_d,k);

			hipMemcpy(top_h,top_d,sizeof(int)*dimx*dimy,hipMemcpyDeviceToHost);
			hipMemcpy(max_h,max_d,sizeof(float)*dimx*dimy,hipMemcpyDeviceToHost);
		}

	
	hipFree(max_d);
	hipFree(var_d);
	hipFree(top_d);

	FILE *topo;
	topo=fopen("Resultados/topo","w+");
	for(i=0;i<dimx*dimy;i++){
		if(i%dimy==0 && i!=0)
			fprintf(topo,"\n");
			fprintf(topo,"%d ",top_h[i]);
	}

	fclose(topo);
	t_GPU=clock();
	t = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("tiempo de procesamiento: %6.3f s\n",t);

}//FIN funci�n main()


//******************leerMatrizVarianza****************

float* leerMatrizVarianza(int d){
	int i;
	char rutavar[]="VARIANZAS/";
	sprintf(rutavar,"%s%d",rutavar,d);

	FILE* archivo;
	archivo=fopen(rutavar,"r") ;

	float *var;
	var=(float *)malloc(sizeof(float)*dimx*dimy);

	for(i=0;i<dimx*dimy;i++)
		fscanf(archivo,"%f",&var[i]);
	fclose(archivo);

	return var;

}