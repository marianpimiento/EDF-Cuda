/*
The code generates a 3D image of a stack of images.

For each image (matrix) calculate the variance at all points, and then create a topography matrix (relief matrix) with
the position (number in the stack) of the image that had the largest variance in a pixel. The same with the color of the
image (RGB matrices).
*/

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include <time.h>

//************Global variables***************



//************** Kernel CUDA *********************
__global__  void EDF(int *R_d, int *G_d, int *B_d, int *Rf_d, int *Gf_d, int *Bf_d, int *topof_d, float *max_d, int d) {

	
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int  idy = threadIdx.y + blockIdx.y*blockDim.y;
	unsigned long long int id = idx + idy*blockDim.x*gridDim.x;
	unsigned long long int dimx = 1040, dimy = 1388, tam_imag = 1388 * 1040, msk = 3, M_d[9], k;
	float X = 0.f, Xprom = 0.f, Y = 0.f, var = 0.f;
	//printf("Run=%lld   ", id);
	//Rf_d[id] = 99;

	if (id<tam_imag) {

		M_d[0] = ((idx == 0 || idy == 0) ? 0 : G_d[id - 1 - dimy]);
		
		if (id - dimy >= 0 && id - dimy < tam_imag)
			M_d[1] = ((idx == 0) ? 0 : G_d[id - dimy]);
		else 
		{
			printf("val=%lld _LINE_=%d\n" , id - dimy , __LINE__);
			return;
		}
		//M_d[1] = ((idx == 0) ? 0 : 10);
		if (id + 1 - dimy >= 0 && id + 1 - dimy < tam_imag)
			M_d[2] = ((idx == 0 || idy == dimy) ? 0 : G_d[id + 1 - dimy]);
		else 
		{
			printf("val=%lld _LINE_=%d\n" , id + 1 - dimy , __LINE__);
			return;
		}
		
		if (id - 1 >= 0 && id - 1 < tam_imag)
			M_d[3] = ((idy == 0) ? 0 : G_d[id - 1]);
		else 
		{
			printf("val=%lld _LINE_=%d\n" , id - 1 , __LINE__);
			return;
		}
		
		if (id  >= 0 && id  < tam_imag)
			M_d[4] = G_d[id];
		else 
		{
			printf("val=%lld _LINE_=%d\n" , id  , __LINE__);
			return;
		}
		if (id + 1  >= 0 && id + 1  < tam_imag)
			M_d[5] = ((idy == dimy) ? 0 : G_d[id + 1]);
		else 
		{
			printf("val=%lld _LINE_=%d\n" , id + 1  , __LINE__);
			return;
		}
		if (id - 1 + dimy  >= 0 && id - 1 + dimy  < tam_imag)
			M_d[6] = ((idx == dimx || idy == 0) ? 0 : G_d[id - 1 + dimy]);
		else 
		{
			printf("val=%lld _LINE_=%d\n" , id - 1 + dimy  , __LINE__);
			return;
		}
		if (id + dimy  >= 0 && id + dimy  < tam_imag)
			M_d[7] = ((idx == dimx) ? 0 : G_d[id + dimy]);
		else 
		{
			printf("val=%lld _LINE_=%d\n" , id + dimy , __LINE__);
			return;
		}
		if (id + 1 + dimy >= 0 && id + 1 + dimy < tam_imag)
			M_d[8] = ((idx == dimx || idy == dimy) ? 0 : G_d[id + 1 + dimy]);
		else 
		{
			printf("val=%lld _LINE_=%d\n" , id + 1 + dimy , __LINE__);
			return;
		}

		for (k = 0;k<msk*msk;k++)
			X += M_d[k];

		Xprom = ((float)X) / (msk*msk);

		for (k = 0;k<msk*msk;k++)
			Y += (Xprom - M_d[k])*(Xprom - M_d[k]);

		var = Y / (msk*msk);

		//syncthreads();
		__syncthreads();
		//hosam

		if (var>max_d[id]) {
			topof_d[id] = d;
			Rf_d[id]=R_d[id];
			Gf_d[id] = G_d[id];
			Bf_d[id] = B_d[id];
			max_d[id] = var;
		}
	}
}


long msk = 3, dimx = 1040, dimy = 1388, tam_imag = 1388 * 1040;
//*****************Main function**********************
int main(int argc, char* argv[]) {

	//***************Variables**************
	int i, j, m, cont, tam_B, init, fin;
	hipError_t cudaStatus;

	init=atoi(argv[1]);
	fin=atoi(argv[2]);
	//init = 5;
	//fin = 5;
	FILE *matrizR, *matrizG, *matrizB;
	int d;
	float t;
	clock_t tinicio, t_GPU;
	tinicio = clock();

	int *topof_h, *R_h, *G_h, *B_h, *Rf_h, *Gf_h, *Bf_h;
	float *max_h;

	int *topof_d, *R_d, *G_d, *B_d, *Rf_d, *Gf_d, *Bf_d;
	float *max_d;

	//************ Malloc in host and device *************** 
	R_h = (int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&R_d, tam_imag * sizeof(int));
	G_h = (int *)malloc(sizeof(int)*tam_imag);
	cudaStatus = hipMalloc((void**)&G_d, tam_imag * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for G_d Line=%d!\n" , __LINE__);
		exit(0);
	}
	B_h = (int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&B_d, tam_imag * sizeof(int));

	Rf_h = (int *)malloc(sizeof(int)*tam_imag);
	cudaStatus = hipMalloc((void**)&Rf_d, tam_imag * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed for Rf_d!\n");
		exit(0);
	}
	Gf_h = (int *)malloc(sizeof(int)*tam_imag);
	cudaStatus=hipMalloc((void**)&Gf_d, tam_imag * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
		}
	Bf_h = (int *)malloc(sizeof(int)*tam_imag);
	cudaStatus=hipMalloc((void**)&Bf_d, tam_imag * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
		}
	topof_h = (int *)malloc(sizeof(int)*tam_imag);
	cudaStatus=hipMalloc((void**)&topof_d, tam_imag * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
		}

	max_h = (float *)malloc(sizeof(float)*tam_imag);
	cudaStatus=hipMalloc((void**)&max_d, tam_imag * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
		}

	//cudaMemset((void *)max_h, 0, sizeof(float)*tam_imag);

	//*************** Principal FOR ****************
	for (d = init;d <= fin;d++) {

		//printf("d=%d \n", d);
		//*****************Read RGB files****************
		char rutaR[1024];
		//sprintf(rutaR, "%s%d%s", "D:/Freelancer/cuda/RGB/", d, "/R.txt");
		sprintf(rutaR, "%s%d%s","RGB/",d,"/R"); 
		matrizR = fopen(rutaR, "r+");
		char rutaG[1024];
		//sprintf(rutaG, "%s%d%s", "D:/Freelancer/cuda/RGB/", d, "/G.txt");
		sprintf(rutaG, "%s%d%s","RGB/",d,"/G"); 
		matrizG = fopen(rutaG, "r+");
		if (!matrizG)
		{
			printf("Error open file \n");
			exit(0);
		}

		char rutaB[1024];
		//sprintf(rutaB, "%s%d%s", "D:/Freelancer/cuda/RGB/", d, "/B.txt");
		sprintf(rutaB, "%s%d%s","RGB/",d,"/B"); 
		matrizB = fopen(rutaB, "r+");
		printf("dimx=%d\n", dimx);
		printf("dimy=%d\n", dimy);
		printf("tam_imag=%d\n", tam_imag);
		printf("dimx*dimy=%d\n", dimx*dimy);


		for (i = 0;i<dimx;i++)
		{
			for (j = 0;j<dimy;j++)
			{
				fscanf(matrizR, "%d", &R_h[i*dimy + j]);
				fscanf(matrizG, "%d", &G_h[i*dimy + j]);
				fscanf(matrizB, "%d", &B_h[i*dimy + j]);
			}
		}
		fclose(matrizR);
		fclose(matrizG);
		fclose(matrizB);

		//***************** Kernel EDF *******************

		cudaStatus = hipMemcpy(R_d, R_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed for %d!\n", __LINE__);
			exit(0);
		}

		cudaStatus = hipMemcpy(G_d, G_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus=hipMemcpy(B_d, B_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		cudaStatus = hipMemcpy(Rf_d, Rf_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus=hipMemcpy(Gf_d, Gf_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus=hipMemcpy(Bf_d, Bf_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		cudaStatus=hipMemcpy(topof_d, topof_h, sizeof(int)*tam_imag, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus=hipMemcpy(max_d, max_h, sizeof(float)*tam_imag, hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}

		dim3 Grid(347, 20);
		dim3 Block(13, 16);

		EDF <<<Grid, Block >>>(R_d, G_d, B_d, Rf_d, Gf_d, Bf_d, topof_d, max_d, d);
		printf("\n\n FINISH \n\n");
		//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ The code stops here
		cudaStatus = hipMemcpy(Rf_h, Rf_d, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
			fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
			exit(0);
		}
		cudaStatus=hipMemcpy(Gf_h, Gf_d, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
		}
		cudaStatus=hipMemcpy(Bf_h, Bf_d, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
		}

		cudaStatus=hipMemcpy(topof_h, topof_d, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
		}
		cudaStatus=hipMemcpy(max_h, max_d, sizeof(float)*tam_imag, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed line %d!\n", __LINE__);
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(cudaStatus));
		exit(0);
		}

	} //End for

	  //****************Save results**************
	printf("Finish\n");
	FILE *archTopo, *archR, *archG, *archB;
	/*archTopo = fopen("D:/Freelancer/cuda/Resultados/topo.txt", "w+");
	archR = fopen("D:/Freelancer/cuda/Resultados/Resultados/R.txt", "w+");
	archG = fopen("D:/Freelancer/cuda/Resultados/Resultados/G.txt", "w+");
	archB = fopen("D:/Freelancer/cuda/Resultados/Resultados/B.txt", "w+");
	*/
	archTopo=fopen("Resultados/topo","w+");
	archR=fopen("Resultados/R","w+");
	archG=fopen("Resultados/G","w+");
	archB=fopen("Resultados/B","w+");
	for (i = 0;i<dimx;i++) {
		for (j = 0;j<dimy;j++) {
			fprintf(archTopo, "%d ", topof_h[i*dimy + j]);
			fprintf(archR, "%d ", Rf_h[i*dimy + j]);
			fprintf(archG, "%d ", Gf_h[i*dimy + j]);
			fprintf(archB, "%d ", Bf_h[i*dimy + j]);
		}
		fprintf(archTopo, "\n");
		fprintf(archR, "\n");
		fprintf(archG, "\n");
		fprintf(archB, "\n");
	}
	fclose(archTopo);
	fclose(archR);
	fclose(archG);
	fclose(archB);

	free(max_h);
	free(topof_h);
	free(R_h);
	free(G_h);
	free(B_h);
	free(Rf_h);
	free(Gf_h);
	free(Bf_h);

	hipFree(max_d);
	hipFree(topof_d);
	hipFree(R_d);
	hipFree(G_d);
	hipFree(B_d);
	hipFree(Rf_d);
	hipFree(Gf_d);
	hipFree(Bf_d);


	t_GPU = clock();
	t = ((float)t_GPU - (float)tinicio) / CLOCKS_PER_SEC;
	printf("\ntiempo de procesamiento de varianzas: %6.3fs\n", t);

	//getchar ();
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;

}//END Main function
