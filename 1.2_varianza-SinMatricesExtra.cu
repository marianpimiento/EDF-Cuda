#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>

//************variables globales***************
int msk=3, dimx=1040, dimy=1388, tam_imag=1388*1040;

//*******************kernel********************

__global__ void kernel (int *Gext_d,float *var_d){

	int i, dimy_ext, id_p, M_d[9], dimy=1388,tam_imag=1388*1040,msk=3;
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int offset=idx + idy*blockDim.x*gridDim.x;
	int id=offset;

	float X=0.f,Xprom=0.f,Y=0.f;
		
	var_d[id]=0;
	dimy_ext=dimy+2;

	if(offset<tam_imag){
		
		offset+=2*idy;
		id_p=offset+(dimy+msk);

		M_d[0]=Gext_d[offset];
		M_d[1]=Gext_d[offset+1];
		M_d[2]=Gext_d[offset+2];
		M_d[3]=Gext_d[id_p-1];
		M_d[4]=Gext_d[id_p];
		M_d[5]=Gext_d[id_p+1];
		M_d[6]=Gext_d[(id_p-1)+dimy_ext];
		M_d[7]=Gext_d[id_p+dimy_ext];
		M_d[8]=Gext_d[(id_p+1)+dimy_ext];

		for(i=0;i<msk*msk;i++)
			X+=M_d[i];
		Xprom=((float)X)/(msk*msk);

		for(i=0;i<msk*msk;i++)
			Y+=(Xprom-M_d[i])*(Xprom-M_d[i]);
		var_d[id]=Y/(msk*msk);

	}
}


//*****************Funcion Main**********************

int main(int argc,char* argv[]){

	//***************Declaracion de variables**************

	int i,j,d,m,cont,tam_ext,init,fin;
	int *Gext_d, *Gext_h;
	//, **G;

	float t, *var_d,*var_h;

	FILE *archV, *matrizG;

	init=atoi(argv[1]);
	fin=atoi(argv[2]);

	clock_t tinicio, t_GPU;
	tinicio=clock();

	tam_ext=(dimx+2)*(dimy+2);

	//*************For que recorre todas las imagenes ************
	for(d=init;d<=fin;d++){

		printf("d=%d \n", d);
		
		Gext_h=(int *)malloc(sizeof(int)*tam_ext);
		hipMalloc((void**)&Gext_d, tam_ext*sizeof(int));
		var_h=(float *)malloc(sizeof(float)*tam_imag);
		hipMalloc((void**)&var_d,tam_imag*sizeof(float));
		

		//*****************Lectura Matriz G****************

		char rutaG[]="MiTesis/";
		sprintf(rutaG, "%s%d%s","RGB/",d,"/G"); 
		matrizG=fopen(rutaG,"r+");

		cont=0;
		for(i=0;i<dimx+2;i++){
			for(j=0;j<dimy+2;j++){
				if (i==0 || j==0 || i==dimx+1 || j==dimy+1){
					Gext_h[cont]=0;
				} else{
					fscanf(matrizG, "%d", &Gext_h[cont]); 
				}
				cont++;
			}
		}
		fclose(matrizG);


		//******************Llamado de kernel*******************

		dim3 Grid(347,20);
		dim3 Block(13,16);

		hipMemcpy(Gext_d,Gext_h,sizeof(int)*tam_ext,hipMemcpyHostToDevice);

		kernel<<<Grid,Block>>>(Gext_d,var_d);

		hipMemcpy(var_h,var_d,sizeof(float)*tam_imag,hipMemcpyDeviceToHost);


		//****************Almacenamiento matriz de Varianza**************

		char rutaV[]="VARIANZAS/";
		sprintf(rutaV, "%s%d", rutaV,d);
		archV=fopen(rutaV,"w+");

		for(m=0;m<tam_imag;m++){
			if(m%dimy==0 && m!=0){
				fprintf(archV,"\n");
			}
			fprintf(archV,"%f ",var_h[m]);
		}
		fclose(archV);

		free(Gext_h);
		free(var_h);
		//free(G);
		hipFree(var_d);
		hipFree(Gext_d);
	}


	t_GPU=clock();
	t = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("\ntiempo de procesamiento de varianzas: %6.3fs\n",t);

	return 0;

}//FIN funcion main()
