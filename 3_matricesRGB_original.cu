#include "hip/hip_runtime.h"
//****librerias****

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>

//****Variables globales****

int N=93, dimx=1920,dimy=2560,tam_imag=1920*2560;

//****Kernel: Funci�n del device****

__global__ void Kernel(int *R_d, int *G_d, int *B_d, int *T_d, int *Rf, int *Gf, int *Bf, int d){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;

	int tam_imag;
	tam_imag=1920*2560;
	if(idx<tam_imag)
		if(T_d[idx]==d){
			Rf[idx]=R_d[idx];
			Gf[idx]=G_d[idx];
			Bf[idx]=B_d[idx];
		}
}

//****Funci�n main()****

int main(int argc,char* argv[]){

	//****declaraci�n de variables para el host y device****

	int i, j, d, cont;
	int *R_h, *R_d, *G_h, *G_d, *B_h, *B_d, *T_d, *T_h;
	int *R, *G, *B;
	int *Rf, *Gf, *Bf;


	FILE *file, *Red, *Green, *Blue;
	FILE *ArchivoR, *ArchivoG, *ArchivoB;

	//****Leer archivo de la matriz topografica****

	file=fopen("Resultados/topo","r+");

	//******matriz R host y device********

	R_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&R_d, tam_imag*sizeof(int));

	//******matriz G host y device********

	G_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&G_d, tam_imag*sizeof(int));

	//******matriz B host y device********

	B_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&B_d, tam_imag*sizeof(int));

	//******matriz topografica device******

	T_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&T_d, tam_imag*sizeof(int));

	//**********matrices finales device*********

	hipMalloc((void**)&Rf, tam_imag*sizeof(int));
	hipMalloc((void**)&Gf, tam_imag*sizeof(int));
	hipMalloc((void**)&Bf, tam_imag*sizeof(int));

	//******matrices Resultados finales******

	R=(int *)malloc(sizeof(int)*tam_imag);
	G=(int *)malloc(sizeof(int)*tam_imag);
	B=(int *)malloc(sizeof(int)*tam_imag);

	//***********c�lculo del tiempo de procesamiento***********

	float t;
	clock_t tinicio, tfinal;
	tinicio=clock();

	//******matriz topografica***********
	cont=0;

	for(i=0;i<dimx;i++)
		for(j=0;j<dimy;j++){
			fscanf(file, "%d", &T_h[cont]);
			cont++;
		}
	fclose(file)

	//******operaciones*******

	for(d=1;d<=N;d++){

		//*******matriz R*********

		char ruta1[]="MiTesis/";
		sprintf(ruta1, "%s%d%s","RGB/",d,"/R");
		Red=fopen(ruta1,"r+");

		for(i=0;i<dimx*dimy;i++)
			fscanf(Red, "%d", &R_h[i]);
		fclose(Red);

		//*******matriz G*********

		char ruta2[]="MiTesis/";
		sprintf(ruta2, "%s%d%s","RGB/",d,"/G");
		Green=fopen(ruta2,"r+");

		for(i=0;i<dimx*dimy;i++)
			fscanf(Green, "%d", &G_h[i]);
		fclose(Green);

		//*******matriz B*********

		char ruta3[]="MiTesis/";
		sprintf(ruta3, "%s%d%s","RGB/",d,"/B");
		Blue=fopen(ruta3,"r+");

		for(i=0;i<dimx*dimy;i++)
			fscanf(Blue, "%d", &B_h[i]);
		fclose(Blue);

		//********copia de variables del Host al Device***********

		hipMemcpy(R_d,R_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(G_d,G_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(B_d,B_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(T_d,T_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);

		//******llamado del kernel********

		Kernel<<<12288,400>>>(R_d, G_d, B_d, T_d, Rf, Gf, Bf, d);

	}//Fin for

	//copia de variables del Device al Host

	hipMemcpy(R, Rf, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
	hipMemcpy(G, Gf, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
	hipMemcpy(B, Bf, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);

	//almacenamiento de las matrices resultantes*******

	ArchivoR=fopen("Resultados/R","w+");
	ArchivoG=fopen("Resultados/G","w+");
	ArchivoB=fopen("Resultados/B","w+");

	for(i=0;i<tam_imag;i++){
		if(i%dimy==0 && i!=0){
			fprintf(ArchivoR,"\n");
			fprintf(ArchivoG,"\n");
			fprintf(ArchivoB,"\n");
		}

		fprintf(ArchivoR,"%d ",R[i]);
		fprintf(ArchivoG,"%d ",G[i]);
		fprintf(ArchivoB,"%d ",B[i]);
	}

	fclose(ArchivoR);
	fclose(ArchivoG);
	fclose(ArchivoB);

	tfinal=clock();
	t = ((float)tfinal-(float)tinicio)/CLOCKS_PER_SEC;
	printf("tiempo de procesamiento:%6.3f s\n",t);

}//Fin funci�n main()
