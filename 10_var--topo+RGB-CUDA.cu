
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

//************variables globales***************
int msk=3, dimx=1040, dimy=1388, tam_imag=1388*1040;

// [i][j] = i*dimy+j

//************** Kernel CUDA *********************
__global__ void Varianza (int *G_d, float *var_d){

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int id = idx + idy*blockDim.x*gridDim.x;

	int M_d[9], i, dimx=1040, dimy=1388, tam_imag=1388*1040, msk=3;
	float X=0.f,Xprom=0.f,Y=0.f;
	var_d[id]=0;
	//printf("prueba\n");

	if(id<tam_imag){
		//M_d[0]=((i<1 || j<1) ? 0:A[i-1][j-1]);
		/*
		M_d[0]=((idx<1 || idy<1) ? 0:G_d[(idx-1)+(idy-1)*blockDim.x*gridDim.x]);
		M_d[1]=((idx<1) ? 0:G_d[(idx-1)+(idy)*blockDim.x*gridDim.x]);
		M_d[2]=((idx<1 || idy>dimy-2) ? 0:G_d[(idx-1)+(idy+1)*blockDim.x*gridDim.x]);
		M_d[3]=((idy<1) ? 0:G_d[(idx)+(idy-1)*blockDim.x*gridDim.x]);
		M_d[4]=G_d[(idx)+(idy)*blockDim.x*gridDim.x];
		M_d[5]=((idy>dimy-2) ? 0:G_d[(idx)+(idy+1)*blockDim.x*gridDim.x]);
		M_d[6]=((idx>dimx-2 || idy<1) ? 0:G_d[(idx+1)+(idy-1)*blockDim.x*gridDim.x]);
		M_d[7]=((idx>dimx-2) ? 0:G_d[(idx+1)+(idy)*blockDim.x*gridDim.x]);
		M_d[8]=((idx>dimx-2 || idy>dimy-1) ? 0:G_d[(idx+1)+(idy+1)*blockDim.x*gridDim.x]);
 		*/
		
		if (idx==0 || idy==0){
			M_d[0]=0;
		}else{
			M_d[0]=G_d[id-1-dimy];
		}
/*
		if ((idx==0)){
			M_d[1]=0;
		}else{
			M_d[1]=G_d[id-dimy];
			//M_d[1]=8;
		}
/*
		if (idx==0 || idy==dimy){
			M_d[2]=0;
		}else{
			M_d[2]=G_d[id+1-dimy];
		}
*/
		if (idy==0){
			M_d[3]=0;
		}else{
			M_d[3]=G_d[id-1];
		}

		M_d[4]=G_d[id];

		if (idy==dimy){
			M_d[5]=0;
		}else{
			M_d[5]=G_d[id+1];
		}
/*
		if (id==dimx || idy==0){
			M_d[6]=0;
		}else{
			M_d[6]=G_d[id-1+dimy];
		}
*//*
		if (idx==dimx){
			M_d[7]=0;
		}else{
			M_d[7]=G_d[id+dimy];
		}
*//*
		if (idx==dimx || idy==dimy){
			M_d[8]=0;
		}else{
			M_d[8]=G_d[id+1+dimy];
		}
*/
		
		//M_d[0]=1;
		M_d[1]=5;
		M_d[2]=8;
		//M_d[3]=1;
		//M_d[4]=1;
		//M_d[5]=1;
		M_d[6]=2;
		M_d[7]=5;
		M_d[8]=4;
		
		for(i=0;i<msk*msk;i++)
			X+=M_d[i];

		Xprom=((float)X)/(msk*msk);

		for(i=0;i<msk*msk;i++)
			Y+=(Xprom-M_d[i])*(Xprom-M_d[i]);
		
		var_d[id]=Y/(msk*msk);

	}
}


//*****************Funcion main**********************
int main(int argc,char* argv[]){

	//***************Declaracion de variables**************
	int i,j,init,fin,d;
	init=atoi(argv[1]);
	fin=atoi(argv[2]);

	//init=1;
	//fin=328;

	FILE *matrizR, *matrizG, *matrizB;
	float t;
	clock_t tinicio, t_GPU;
	tinicio=clock();

	int *topof_h, *R_h, *G_h, *B_h, *Rf_h, *Gf_h, *Bf_h;
	float *max_h, *var_h;

	int *topof_d, *R_d, *G_d, *B_d, *Rf_d, *Gf_d, *Bf_d;
	float *max_d, *var_d;
	
	//************Inicializacion de variables en el host y en el device *************** 
	
	/* // Declaracion tipo MATRIZ
	max_h=(float **)malloc(sizeof(float)*dimx);
	topof_h=(int **)malloc(sizeof(int)*dimx);
	R_h=(int **)malloc(sizeof(int)*dimx);
	G_h=(int **)malloc(sizeof(int)*dimx);
	B_h=(int **)malloc(sizeof(int)*dimx);
	Rf_h=(int **)malloc(sizeof(int)*dimx);
	Gf_h=(int **)malloc(sizeof(int)*dimx);
	Bf_h=(int **)malloc(sizeof(int)*dimx);
	for(i=0;i<dimx;i++){
		max_h[i]=(float*)malloc(sizeof(float)*dimy);
		topof_h[i]=(int*)malloc(sizeof(int)*dimy);
		R_h[i]=(int*)malloc(sizeof(int)*dimy);
		G_h[i]=(int*)malloc(sizeof(int)*dimy);
		B_h[i]=(int*)malloc(sizeof(int)*dimy);
		Rf_h[i]=(int*)malloc(sizeof(int)*dimy);
		Gf_h[i]=(int*)malloc(sizeof(int)*dimy);
		Bf_h[i]=(int*)malloc(sizeof(int)*dimy);
	}
	var_h=(float *)malloc(sizeof(float)*tam_imag);
	*/

	R_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&R_d, tam_imag*sizeof(int));
	G_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&G_d, tam_imag*sizeof(int));
	B_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&B_d, tam_imag*sizeof(int));
	Rf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Rf_d, tam_imag*sizeof(int));
	Gf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Gf_d, tam_imag*sizeof(int));
	Bf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Bf_d, tam_imag*sizeof(int));
	topof_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&topof_d, tam_imag*sizeof(int));

	max_h=(float *)malloc(sizeof(float)*tam_imag);
	hipMalloc((void**)&max_d, tam_imag*sizeof(float));
	var_h=(float *)malloc(sizeof(float)*tam_imag);
	hipMalloc((void**)&var_d,tam_imag*sizeof(float));

	
	//*************** For cálculo EDF ****************
	for(d=init;d<=fin;d++){

		printf("d=%d \n", d);
		
		//*****************Lecura de matrices RGB en el host****************

		char rutaR[]="";
		sprintf(rutaR, "%s%d%s","RGB/",d,"/R"); 
		matrizR=fopen(rutaR,"r+");

		char rutaG[]="";
		sprintf(rutaG, "%s%d%s","RGB/",d,"/G"); 
		matrizG=fopen(rutaG,"r+");

		char rutaB[]="";
		sprintf(rutaB, "%s%d%s","RGB/",d,"/B"); 
		matrizB=fopen(rutaB,"r+");

		for(i=0;i<dimx;i++){
			for(j=0;j<dimy;j++){
				fscanf(matrizR, "%d", &R_h[i*dimy+j]);
				fscanf(matrizG, "%d", &G_h[i*dimy+j]); 
				fscanf(matrizB, "%d", &B_h[i*dimy+j]); 
			}
		}
		fclose(matrizR);
		fclose(matrizG);
		fclose(matrizB);

		//***************** Kernel Varianza *******************

		hipMemcpy(G_d,G_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);

		dim3 Grid(347,20);
		dim3 Block(13,16);

		Varianza<<<Grid,Block>>>(B_d,var_d);

		printf("Despues de kernel \n");

		hipMemcpy(var_h,var_d,sizeof(float)*tam_imag,hipMemcpyDeviceToHost);
		
		printf("Despues de resultado a host \n");
		//***************** Kernel Varianza *******************
		/*
		cudaMemcpy(R_d,R_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);
		cudaMemcpy(G_d,G_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);
		cudaMemcpy(B_d,B_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);

		cudaMemcpy(Rf_d,Rf_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);
		cudaMemcpy(Gf_d,Gf_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);
		cudaMemcpy(Bf_d,Bf_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);

		cudaMemcpy(topof_d,topof_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);
		cudaMemcpy(max_d,max_h,sizeof(float)*tam_imag,cudaMemcpyHostToDevice);
		cudaMemcpy(var_d,var_h,sizeof(float)*tam_imag,cudaMemcpyHostToDevice);

		dim3 Grid(347,20);
		dim3 Block(13,16);

		TopoRGB<<<Grid,Block>>>(R_d,G_d,B_d,Rf_d,Gf_d,Bf_d,topof_d,max_d,var_d);

		cudaMemcpy(Rf_h,Rf_d,sizeof(int)*tam_imag,cudaMemcpyDeviceToHost);
		cudaMemcpy(Gf_h,Gf_d,sizeof(int)*tam_imag,cudaMemcpyDeviceToHost);
		cudaMemcpy(Bf_h,Bf_d,sizeof(int)*tam_imag,cudaMemcpyDeviceToHost);

		cudaMemcpy(topof_h,topof_d,sizeof(int)*tam_imag,cudaMemcpyDeviceToHost);
		cudaMemcpy(max_h,max_d,sizeof(float)*tam_imag,cudaMemcpyDeviceToHost);

		*/
		//*********************Calculo de TODO ********************
		
	} //Finaliza For cálculo EDF
	printf("***Sale del for \n");

	/*
	// ***************** Generacion de archivos de resultados ************************
	FILE *archTopo, *archR, *archG, *archB;
	archTopo=fopen("Resultados/topos10","w+");
	archR=fopen("Resultados/R10","w+");
	archG=fopen("Resultados/G10","w+");
	archB=fopen("Resultados/B10","w+");
	for(i=0;i<dimx;i++){
		for(j=0;j<dimy;j++){
			fprintf(archTopo,"%d ",topof_h[i*dimy+j]);
			fprintf(archR,"%d ",Rf_h[i*dimy+j]);
			fprintf(archG,"%d ",Gf_h[i*dimy+j]);
			fprintf(archB,"%d ",Bf_h[i*dimy+j]);
		}
		fprintf(archTopo,"\n");
		fprintf(archR,"\n");
		fprintf(archG,"\n");
		fprintf(archB,"\n");
	}
	fclose(archTopo);
	fclose(archR);
	fclose(archG);
	fclose(archB);
	*/

	//***************** Archivo de varianza final
	FILE *archVar;
	archVar=fopen("Resultados/VarUltima","w+");
	for(i=0;i<dimx;i++){
		for(j=0;j<dimy;j++){
			fprintf(archVar,"%f ",var_h[i*dimy+j]);

		}
		fprintf(archVar,"\n");
	}
	fclose(archVar);


	free(var_h);
	free(max_h);
	free(topof_h);
	free(R_h);
	free(G_h);
	free(B_h);
	free(Rf_h);
	free(Gf_h);
	free(Bf_h);

	hipFree(var_d);
	hipFree(max_d);
	hipFree(topof_d);
	hipFree(R_d);
	hipFree(G_d);
	hipFree(B_d);
	hipFree(Rf_d);
	hipFree(Gf_d);
	hipFree(Bf_d);

	
	t_GPU=clock();
	t = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("\ntiempo de procesamiento de varianzas: %6.3fs\n",t);

	//getchar ();
	return 0;

}//FIN funcion main()
