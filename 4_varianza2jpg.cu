#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>

//************variables globales***************
int msk=3, dimx=1388, dimy=1040, tam_imag=1388*1040;

//*******************kernel********************

__global__ void kernel (int *B_d,float *var_d){

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int offset=idx + idy*blockDim.x*gridDim.x;

	int id=offset;
	int i;
	float X=0.f,Xprom=0.f,Y=0.f;
	int dimy=1040,tam_imag=1388*1040,msk=3;
	var_d[id]=0;

	if(offset<tam_imag){
		int dimy_B=dimy+2;

		offset+=2*idy;
		int id_p=offset+(dimy+msk);

		int M_d[9];

		M_d[0]=B_d[offset];
		M_d[1]=B_d[offset+1];
		M_d[2]=B_d[offset+2];
		M_d[3]=B_d[id_p-1];
		M_d[4]=B_d[id_p];
		M_d[5]=B_d[id_p+1];
		M_d[6]=B_d[(id_p-1)+dimy_B];
		M_d[7]=B_d[id_p+dimy_B];
		M_d[8]=B_d[(id_p+1)+dimy_B];

		for(i=0;i<msk*msk;i++)
			X+=M_d[i];
		Xprom=((float)X)/(msk*msk);

		for(i=0;i<msk*msk;i++)
			Y+=(Xprom-M_d[i])*(Xprom-M_d[i]);
		var_d[id]=Y/(msk*msk);

	}
}


//*****************funcion main**********************

int main(int argc,char* argv[]){

	//***************declaracion de variables**************

	printf("\nInicia main");
	int i,j,m,cont,tam_B, init,fin;
	init=atoi(argv[1]);
	fin=atoi(argv[2]);

	tam_B=(dimx+2)*(dimy+2);

	printf("\nCrea FILE\n");
	FILE *arch, *matrizG, *archM; //+++++++++++++++++++++++++++++++++++++++++++++++++++++++++

	int **A;
	int B[dimx+2][dimy+2];

	float t;
	clock_t tinicio, t_GPU;
	tinicio=clock();

	int *B_d, *B_h;
	float *var_d,*var_h;

	int d;
	for(d=init;d<=fin;d++){

		//*******************declaracion de variables***************
		printf("d=%d \n", d);
		
		B_h=(int *)malloc(sizeof(int)*tam_B);
		hipMalloc((void**)&B_d, tam_B*sizeof(int));
		var_h=(float *)malloc(sizeof(float)*tam_imag);
		hipMalloc((void**)&var_d,tam_imag*sizeof(float));
		
		A=(int **)malloc(sizeof(int)*dimx);
		for(i=0;i<dimx;i++)
			A[i]=(int*)malloc(sizeof(int)*dimy);

		//*****************calculo matriz B****************

		char ruta1[]="MiTesis/";
		sprintf(ruta1, "%s%d%s","RGB/",d,"/G2.jpg");  //++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
		matrizG=fopen(ruta1,"r+");

		for(i=0;i<dimx;i++){
			for(j=0;j<dimy;j++){
				fscanf(matrizG, "%d", &A[i][j]); 
			}
		}
		fclose(matrizG);

		//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

		char rutaM[]="matrizP/";
		sprintf(rutaM, "%s%d", rutaM,d);
		archM=fopen(rutaM,"w+");

		for(i=0;i<dimx;i++){
			fprintf(archM,"\n");
			for(j=0;j<dimy;j++){
				fprintf(archM,"%d ",A[i][j]);
			}
		}
		fclose(archM);

		//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

		cont=0;
		for(i=0;i<dimx+2;i++){
			//printf("\n");
			for(j=0;j<dimy+2;j++){
				B[i][j]=((i==0 || j==0 || i==dimx+1 || j==dimy+1) ? 0:A[i-1][j-1]);
				B_h[cont]=B[i][j];
				cont++;
			}
		}

		//******************llamado de kernel*******************

		dim3 Grid(347,20);
		dim3 Block(13,16);

		hipMemcpy(B_d,B_h,sizeof(int)*tam_B,hipMemcpyHostToDevice);

		kernel<<<Grid,Block>>>(B_d,var_d);

		hipMemcpy(var_h,var_d,sizeof(float)*tam_imag,hipMemcpyDeviceToHost);


		//****************almacenamiento matriz de varianza**************

		char rutaV[]="VARIANZAS/";
		sprintf(rutaV, "%s%dj", rutaV,d); //++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
		arch=fopen(rutaV,"w+");

		for(m=0;m<tam_imag;m++){
			if(m%dimy==0 && m!=0){
				fprintf(arch,"\n");
			}
			fprintf(arch,"%f ",var_h[m]); // "%.2f " - Imprimiria 2 decimales
		}
		fclose(arch);

		free(B_h);
		free(var_h);
		free(A);
		hipFree(var_d);
		hipFree(B_d);
	}


	t_GPU=clock();
	t = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("\ntiempo de procesamiento de varianzas: %6.3fs\n",t);

	return 0;

}//FIN funcion main()
