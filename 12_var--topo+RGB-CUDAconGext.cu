#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>

//************variables globales***************
int msk=3, dimx=1040, dimy=1388, tam_imag=1388*1040;

//*******************kernel********************

__global__ void varianza (int *Gext_d,float *var_d){

	int i, dimy_ext, id_p, M_d[9], dimy=1388,tam_imag=1388*1040,msk=3;
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int offset=idx + idy*blockDim.x*gridDim.x;
	int id=offset;

	float X=0.f,Xprom=0.f,Y=0.f;
		
	//float var=0;
	//var_d[id]=0;
	
	if(offset<tam_imag){
		
		dimy_ext=dimy+2;
		offset+=2*idy;
		id_p=offset+(dimy+msk);

		M_d[0]=Gext_d[offset];
		M_d[1]=Gext_d[offset+1];
		M_d[2]=Gext_d[offset+2];
		M_d[3]=Gext_d[id_p-1];
		M_d[4]=Gext_d[id_p];
		M_d[5]=Gext_d[id_p+1];
		M_d[6]=Gext_d[(id_p-1)+dimy_ext];
		M_d[7]=Gext_d[id_p+dimy_ext];
		M_d[8]=Gext_d[(id_p+1)+dimy_ext];

		for(i=0;i<msk*msk;i++)
			X+=M_d[i];
		Xprom=((float)X)/(msk*msk);

		for(i=0;i<msk*msk;i++)
			Y+=(Xprom-M_d[i])*(Xprom-M_d[i]);
		
		//var=Y/(msk*msk);
		var_d[id]=Y/(msk*msk);

	}
}


__global__ void topografia (float *var_d,int *topof_d,float *max_d, int d){

	int idx=threadIdx.x + blockIdx.x*blockDim.x;
	int tam_imag=1388*1040;

	if(idx<tam_imag){
		if(var_d[idx]>max_d[idx]){
			topof_d[idx]=d;
			max_d[idx]=var_d[idx];
			/*	Rf_d[id]=R_d[id];
			Gf_d[id]=G_d[id];
			Bf_d[id]=B_d[id];*/
		}
	}
}

//*****************Funcion Main**********************

int main(int argc,char* argv[]){

	//***************Declaracion de variables**************

	int i,j,d,m,cont,tam_ext,init,fin;

	init=atoi(argv[1]);
	fin=atoi(argv[2]);

	FILE *matrizR, *matrizG, *matrizB, *matrizGext;
	
	float t;
	clock_t tinicio, t_GPU;
	tinicio=clock();

	tam_ext=(dimx+2)*(dimy+2);

	int *topof_h, *R_h, *G_h, *B_h, *Rf_h, *Gf_h, *Bf_h, *Gext_h;
	float *max_h, *var_h;

	int *topof_d, *R_d, *G_d, *B_d, *Rf_d, *Gf_d, *Bf_d, *Gext_d;
	float *max_d, *var_d;


	//************Inicializacion de variables en el host y en el device *************** 
	
	R_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&R_d, tam_imag*sizeof(int));
	G_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&G_d, tam_imag*sizeof(int));
	B_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&B_d, tam_imag*sizeof(int));
	Rf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Rf_d, tam_imag*sizeof(int));
	Gf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Gf_d, tam_imag*sizeof(int));
	Bf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Bf_d, tam_imag*sizeof(int));
	topof_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&topof_d, tam_imag*sizeof(int));

	Gext_h=(int *)malloc(sizeof(int)*tam_ext);
	hipMalloc((void**)&Gext_d, tam_ext*sizeof(int));

	max_h=(float *)malloc(sizeof(float)*tam_imag);
	hipMalloc((void**)&max_d, tam_imag*sizeof(float));
	
	//cudaMemset((void *) max_d, 0, sizeof(float)*tam_imag);
	//void *memset(void *str, int c, size_t n)
	//memset((void *) max_h, 0, sizeof(float)*tam_imag);

	for(i=0;i<tam_imag;i++){
		max_h[i]=0.0;
		topof_h[i]=0;
	}

	printf("Antes for principal\n");

	//*************For que recorre todas las imagenes ************
	for(d=init;d<=fin;d++){

		printf("d=%d \n", d);

		var_h=(float *)malloc(sizeof(float)*tam_imag);
		hipMalloc((void**)&var_d,tam_imag*sizeof(float));

		for(i=0;i<tam_imag;i++){
			var_h[i]=0;
		}
		
		//*****************Lecura de matrices RGB en el host****************
/*
		char rutaR[]="";
		sprintf(rutaR, "%s%d%s","RGB/",d,"/R"); 
		matrizR=fopen(rutaR,"r+");

		char rutaG[]="";
		sprintf(rutaG, "%s%d%s","RGB/",d,"/G"); 
		matrizG=fopen(rutaG,"r+");

		char rutaB[]="";
		sprintf(rutaB, "%s%d%s","RGB/",d,"/B"); 
		matrizB=fopen(rutaB,"r+");

		for(i=0;i<dimx;i++){
			for(j=0;j<dimy;j++){
				fscanf(matrizR, "%d", &R_h[i*dimy+j]);
				fscanf(matrizG, "%d", &G_h[i*dimy+j]); 
				fscanf(matrizB, "%d", &B_h[i*dimy+j]); 
			}
		}
		fclose(matrizR);
		fclose(matrizG);
		fclose(matrizB);
*/
		//G extendido

		char rutaGext[]="";
		sprintf(rutaGext, "%s%d%s","RGB/",d,"/G"); 
		matrizGext=fopen(rutaGext,"r+");

		cont=0;
		for(i=0;i<dimx+2;i++){
			for(j=0;j<dimy+2;j++){
				if (i==0 || j==0 || i==dimx+1 || j==dimy+1){
					Gext_h[cont]=0;
				} else{
					fscanf(matrizGext, "%d", &Gext_h[cont]); 
				}
				cont++;
			}
		}
		fclose(matrizGext);

		printf("Despues lectura matrices \n");


		//******************Llamado kernel varianza*******************  ++++++++++++++++++++++++++++++++++++
		printf("*Kenel varianza \n");
		hipMemcpy(Gext_d,Gext_h,sizeof(int)*tam_ext,hipMemcpyHostToDevice);

		printf("Despues copia a device\n");

		dim3 Grid(347,20);
		dim3 Block(13,16);

		varianza<<<Grid,Block>>>(Gext_d,var_d);
		printf("Despues kernel \n");

		hipMemcpy(var_h,var_d,sizeof(float)*tam_imag,hipMemcpyDeviceToHost);
		printf("Despues copia a host\n");
		printf("var_h[0]= %f\n", var_h[0]);


		//******************Llamado kernel topografia******************* ++++++++++++++++++++++++++++++++
		printf("*Kenel topografia \n");
/*
		cudaMemcpy(R_d,R_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);
		cudaMemcpy(G_d,G_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);
		cudaMemcpy(B_d,B_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);

		cudaMemcpy(Rf_d,Rf_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);
		cudaMemcpy(Gf_d,Gf_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);
		cudaMemcpy(Bf_d,Bf_h,sizeof(int)*tam_imag,cudaMemcpyHostToDevice);
*/
		hipMemcpy(var_d,var_h,sizeof(float)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(topof_d,topof_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(max_d,max_h,sizeof(float)*tam_imag,hipMemcpyHostToDevice);

		printf("Despues copia a device\n");

		//dim3 Grid(347,20);
		//dim3 Block(13,16);

		//topografia<<<6940,208>>>(R_d,G_d,B_d,Rf_d,Gf_d,Bf_d,topof_d,max_d,var_d,d);
		topografia<<<6940,208>>>(var_d,topof_d,max_d,d);
		printf("Despues kernel \n");

/*		cudaMemcpy(Rf_h,Rf_d,sizeof(int)*tam_imag,cudaMemcpyDeviceToHost);
		cudaMemcpy(Gf_h,Gf_d,sizeof(int)*tam_imag,cudaMemcpyDeviceToHost);
		cudaMemcpy(Bf_h,Bf_d,sizeof(int)*tam_imag,cudaMemcpyDeviceToHost);
*/
		hipMemcpy(topof_h,topof_d,sizeof(int)*tam_imag,hipMemcpyDeviceToHost);
		hipMemcpy(max_h,max_d,sizeof(float)*tam_imag,hipMemcpyDeviceToHost);
		printf("Despues copia a host\n");
		printf("topof_h[0]= %d\n", topof_h[0]);

	}//Finaliza For principal

	//****************Almacenamiento matrices**************

	FILE *archTopo, *archR, *archG, *archB, *archV;
	archTopo=fopen("Resultados/topos12","w+");
	/*archR=fopen("Resultados/R12","w+");
	archG=fopen("Resultados/G12","w+");
	archB=fopen("Resultados/B12","w+");
	archV=fopen("Resultados/VarUltima","w+");*/
	for(i=0;i<dimx;i++){
		for(j=0;j<dimy;j++){
			fprintf(archTopo,"%d ",topof_h[i*dimy+j]);
			/*fprintf(archR,"%d ",Rf_h[i*dimy+j]);
			fprintf(archG,"%d ",Gf_h[i*dimy+j]);
			fprintf(archB,"%d ",Bf_h[i*dimy+j]);
			fprintf(archV,"%f ",var_h[i*dimy+j]);*/
		}
		fprintf(archTopo,"\n");
		/*fprintf(archR,"\n");
		fprintf(archG,"\n");
		fprintf(archB,"\n");
		fprintf(archV,"\n");*/
	}
	fclose(archTopo);
/*	fclose(archR);
	fclose(archG);
	fclose(archB);
	fclose(archV);*/


	//****************Libera memoria**************
	free(R_h);
	hipFree(R_d);
	free(G_h);
	hipFree(G_d);
	free(B_h);
	hipFree(B_d);
	free(Rf_h);
	hipFree(Rf_d);
	free(Gf_h);
	hipFree(Gf_d);
	free(Bf_h);
	hipFree(Bf_d);
	free(Gext_h);
	hipFree(Gext_d);
	free(topof_h);
	hipFree(topof_d);
	free(max_h);
	hipFree(max_d);

	t_GPU=clock();
	t = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("\ntiempo de procesamiento de varianzas: %6.3fs\n",t);

	return 0;

}//FIN funcion main()
