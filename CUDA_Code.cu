#include "hip/hip_runtime.h"
/*
The code generates a 3D image of a stack of images.

For each image (matrix) calculate the variance at all points, and then create a topography matrix (relief matrix) with 
the position (number in the stack) of the image that had the largest variance in a pixel. The same with the color of the 
image (RGB matrices).
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

//************Global variables***************
int msk=3, dimx=1040, dimy=1388, tam_imag=1388*1040;


//************** Kernel CUDA *********************
__global__ void EDF(int *R_d,int *G_d,int *B_d,int *Rf_d,int *Gf_d,int *Bf_d,int *topof_d,float *max_d, int d){

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int id = idx + idy*blockDim.x*gridDim.x;

	int dimx=1040, dimy=1388, tam_imag=1388*1040, msk=3, M_d[9], k;
	float X=0.f,Xprom=0.f,Y=0.f, var=0.f;

	if(id<tam_imag){

		M_d[0]=((idx==0 || idy==0) ? 0:G_d[id-1-dimy]);
		M_d[1]=((idx==0) ? 0:G_d[id-dimy]);
		M_d[2]=((idx==0 || idy==dimy) ? 0:G_d[id+1-dimy]);
		M_d[3]=((idy==0) ? 0:G_d[id-1]);
		M_d[4]=G_d[id];
		M_d[5]=((idy==dimy) ? 0:G_d[id+1]);
		M_d[6]=((idx==dimx || idy==0) ? 0:G_d[id-1+dimy]);
		M_d[7]=((idx==dimx) ? 0:G_d[id+dimy]);
		M_d[8]=((idx==dimx || idy==dimy) ? 0:G_d[id+1+dimy]);

		for(k=0;k<msk*msk;k++)
			X+=M_d[k];
				
		Xprom=((float)X)/(msk*msk);

		for(k=0;k<msk*msk;k++)
			Y+=(Xprom-M_d[k])*(Xprom-M_d[k]);

		var=Y/(msk*msk);

		syncthreads();

		if(var>max_d[id]){
			topof_d[id]=d;
			Rf_d[id]=R_d[id];
			Gf_d[id]=G_d[id];
			Bf_d[id]=B_d[id];
			max_d[id]=var;
		}
	}
}



//*****************Main function**********************
int main(int argc,char* argv[]){

	//***************Variables**************
	int i,j,m,cont,tam_B, init,fin;
	
	init=atoi(argv[1]);
	fin=atoi(argv[2]);

	FILE *matrizR, *matrizG, *matrizB;
	int d;
	float t;
	clock_t tinicio, t_GPU;
	tinicio=clock();

	int *topof_h, *R_h, *G_h, *B_h, *Rf_h, *Gf_h, *Bf_h;
	float *max_h;

	int *topof_d, *R_d, *G_d, *B_d, *Rf_d, *Gf_d, *Bf_d;
	float *max_d;
	
	//************ Malloc in host and device *************** 
	
	R_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&R_d, tam_imag*sizeof(int));
	G_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&G_d, tam_imag*sizeof(int));
	B_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&B_d, tam_imag*sizeof(int));
	Rf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Rf_d, tam_imag*sizeof(int));
	Gf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Gf_d, tam_imag*sizeof(int));
	Bf_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&Bf_d, tam_imag*sizeof(int));
	topof_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&topof_d, tam_imag*sizeof(int));

	max_h=(float *)malloc(sizeof(float)*tam_imag);
	hipMalloc((void**)&max_d, tam_imag*sizeof(float));

	hipMemset((void *) max_h, 0, sizeof(float)*tam_imag);
	
	//*************** Principal FOR ****************
	for(d=init;d<=fin;d++){

		printf("d=%d \n", d);
		
		//*****************Read RGB files****************

		char rutaR[]="";
		sprintf(rutaR, "%s%d%s","RGB/",d,"/R"); 
		matrizR=fopen(rutaR,"r+");

		char rutaG[]="";
		sprintf(rutaG, "%s%d%s","RGB/",d,"/G"); 
		matrizG=fopen(rutaG,"r+");

		char rutaB[]="";
		sprintf(rutaB, "%s%d%s","RGB/",d,"/B"); 
		matrizB=fopen(rutaB,"r+");

		for(i=0;i<dimx;i++){
			for(j=0;j<dimy;j++){
				fscanf(matrizR, "%d", &R_h[i*dimy+j]);
				fscanf(matrizG, "%d", &G_h[i*dimy+j]); 
				fscanf(matrizB, "%d", &B_h[i*dimy+j]); 
			}
		}
		fclose(matrizR);
		fclose(matrizG);
		fclose(matrizB);

		//***************** Kernel EDF *******************

		hipMemcpy(R_d,R_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(G_d,G_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(B_d,B_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);

		hipMemcpy(Rf_d,Rf_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(Gf_d,Gf_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(Bf_d,Bf_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);

		hipMemcpy(topof_d,topof_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(max_d,max_h,sizeof(float)*tam_imag,hipMemcpyHostToDevice);

		dim3 Grid(347,20);
		dim3 Block(13,16);

		EDF<<<Grid,Block>>>(R_d,G_d,B_d,Rf_d,Gf_d,Bf_d,topof_d,max_d,d);

		//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ The code stops here
		hipMemcpy(Rf_h,Rf_d,sizeof(int)*tam_imag,hipMemcpyDeviceToHost);
		hipMemcpy(Gf_h,Gf_d,sizeof(int)*tam_imag,hipMemcpyDeviceToHost);
		hipMemcpy(Bf_h,Bf_d,sizeof(int)*tam_imag,hipMemcpyDeviceToHost);

		hipMemcpy(topof_h,topof_d,sizeof(int)*tam_imag,hipMemcpyDeviceToHost);
		hipMemcpy(max_h,max_d,sizeof(float)*tam_imag,hipMemcpyDeviceToHost);

	} //End for

	//****************Save results**************

	FILE *archTopo, *archR, *archG, *archB;
	archTopo=fopen("Resultados/topo","w+");
	archR=fopen("Resultados/R","w+");
	archG=fopen("Resultados/G","w+");
	archB=fopen("Resultados/B","w+");
	for(i=0;i<dimx;i++){
		for(j=0;j<dimy;j++){
			fprintf(archTopo,"%d ",topof_h[i*dimy+j]);
			fprintf(archR,"%d ",Rf_h[i*dimy+j]);
			fprintf(archG,"%d ",Gf_h[i*dimy+j]);
			fprintf(archB,"%d ",Bf_h[i*dimy+j]);
		}
		fprintf(archTopo,"\n");
		fprintf(archR,"\n");
		fprintf(archG,"\n");
		fprintf(archB,"\n");
	}
	fclose(archTopo);
	fclose(archR);
	fclose(archG);
	fclose(archB);

	free(max_h);
	free(topof_h);
	free(R_h);
	free(G_h);
	free(B_h);
	free(Rf_h);
	free(Gf_h);
	free(Bf_h);

	hipFree(max_d);
	hipFree(topof_d);
	hipFree(R_d);
	hipFree(G_d);
	hipFree(B_d);
	hipFree(Rf_d);
	hipFree(Gf_d);
	hipFree(Bf_d);

	
	t_GPU=clock();
	t = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
	printf("\ntiempo de procesamiento de varianzas: %6.3fs\n",t);

	//getchar ();
	return 0;

}//END Main function
