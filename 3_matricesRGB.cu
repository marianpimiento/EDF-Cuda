#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>

//****Variables globales****
int dimx=1040, dimy=1388,tam_imag=1388*1040;

//****Kernel: Funcion del device****

__global__ void Kernel(int *R_d, int *G_d, int *B_d, int *T_d, int *Rf, int *Gf, int *Bf, int d){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;

	int tam_imag;
	tam_imag=1388*1040;
	if(idx<tam_imag)
		if(T_d[idx]==d){
			Rf[idx]=R_d[idx];
			Gf[idx]=G_d[idx];
			Bf[idx]=B_d[idx];
		}
}

//****Funcion main()****

int main(int argc,char* argv[]){

	//****declaracion de variables para el host y device****
	int N=atoi(argv[2]);
	int i, j, d, cont;
	int *R_h, *R_d, *G_h, *G_d, *B_h, *B_d, *T_d, *T_h;
	int *R, *G, *B;
	int *Rf, *Gf, *Bf;


	FILE *file, *Red, *Green, *Blue;
	FILE *ArchivoR, *ArchivoG, *ArchivoB;

	//****Leer archivo de la matriz topografica****

	file=fopen("Resultados/topo","r+");

	//******matriz R host y device********

	R_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&R_d, tam_imag*sizeof(int));

	//******matriz G host y device********

	G_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&G_d, tam_imag*sizeof(int));

	//******matriz B host y device********

	B_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&B_d, tam_imag*sizeof(int));

	//******matriz topografica device******

	T_h=(int *)malloc(sizeof(int)*tam_imag);
	hipMalloc((void**)&T_d, tam_imag*sizeof(int));

	//**********matrices finales device*********

	hipMalloc((void**)&Rf, tam_imag*sizeof(int));
	hipMalloc((void**)&Gf, tam_imag*sizeof(int));
	hipMalloc((void**)&Bf, tam_imag*sizeof(int));

	//******matrices Resultados finales******

	R=(int *)malloc(sizeof(int)*tam_imag);
	G=(int *)malloc(sizeof(int)*tam_imag);
	B=(int *)malloc(sizeof(int)*tam_imag);

	//***********calculo del tiempo de procesamiento***********

	float t;
	clock_t tinicio, tfinal;
	tinicio=clock();

	//******matriz topografica***********
	cont=0;

	for(i=0;i<dimx;i++)
		for(j=0;j<dimy;j++){
			fscanf(file, "%d", &T_h[cont]);
			cont++;
		}
	fclose(file);

	//******operaciones*******

	for(d=1;d<=N;d++){

		//*******matriz R*********
		printf("d=%d\n", d);
		char ruta1[]="MiTesis/";
		sprintf(ruta1, "%s%d%s","RGB/",d,"/R");
		Red=fopen(ruta1,"r+");

		for(i=0;i<dimx*dimy;i++)
			fscanf(Red, "%d", &R_h[i]);
		fclose(Red);

		//*******matriz G*********

		char ruta2[]="MiTesis/";
		sprintf(ruta2, "%s%d%s","RGB/",d,"/G");
		Green=fopen(ruta2,"r+");

		for(i=0;i<dimx*dimy;i++)
			fscanf(Green, "%d", &G_h[i]);
		fclose(Green);

		//*******matriz B*********

		char ruta3[]="MiTesis/";
		sprintf(ruta3, "%s%d%s","RGB/",d,"/B");
		Blue=fopen(ruta3,"r+");

		for(i=0;i<dimx*dimy;i++)
			fscanf(Blue, "%d", &B_h[i]);
		fclose(Blue);

		//********copia de variables del Host al Device***********

		hipMemcpy(R_d,R_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(G_d,G_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(B_d,B_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);
		hipMemcpy(T_d,T_h,sizeof(int)*tam_imag,hipMemcpyHostToDevice);

		//******llamado del kernel********

		Kernel<<<6940,208>>>(R_d, G_d, B_d, T_d, Rf, Gf, Bf, d);

	}//Fin for

	//copia de variables del Device al Host

	hipMemcpy(R, Rf, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
	hipMemcpy(G, Gf, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);
	hipMemcpy(B, Bf, sizeof(int)*tam_imag, hipMemcpyDeviceToHost);

	//almacenamiento de las matrices resultantes*******

	ArchivoR=fopen("Resultados/R","w+");
	ArchivoG=fopen("Resultados/G","w+");
	ArchivoB=fopen("Resultados/B","w+");

	for(i=0;i<tam_imag;i++){
		if(i%dimy==0 && i!=0){
			fprintf(ArchivoR,"\n");
			fprintf(ArchivoG,"\n");
			fprintf(ArchivoB,"\n");
		}

		fprintf(ArchivoR,"%d ",R[i]);
		fprintf(ArchivoG,"%d ",G[i]);
		fprintf(ArchivoB,"%d ",B[i]);
	}

	fclose(ArchivoR);
	fclose(ArchivoG);
	fclose(ArchivoB);

	tfinal=clock();
	t = ((float)tfinal-(float)tinicio)/CLOCKS_PER_SEC;
	printf("tiempo de procesamiento de RGB:%6.3f s\n",t);

}//Fin funcion main()
