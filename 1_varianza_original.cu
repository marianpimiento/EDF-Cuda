#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>

//************variables globales***************

int msk=3, dimx=1920, dimy=2560, tam_imag=1920*2560;

//*******************kernel********************

__global__ void kernel (int *B_d,float *var_d){

	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idy = threadIdx.y + blockIdx.y*blockDim.y;
	int offset=idx + idy*blockDim.x*gridDim.x;

	int id=offset;
	int I;
	float X=0.f,Xprom=0.f,Y=0.f;
	int dimy=2560,tam_imag=1920*2560,msk=3;
	var_d[id]=0;

	if(offset<tam_imag){
		int dimy_B=dimy+2;

		offset+=2*idy;
		int id_p=offset+(dimy+msk);

		int M_d[9];

		M_d[0]=B_d[offset];
		M_d[1]=B_d[offset+1];
		M_d[2]=B_d[offset+2];
		M_d[3]=B_d[id_p-1];
		M_d[4]=B_d[id_p];
		M_d[5]=B_d[id_p+1];
		M_d[6]=B_d[(id_p-1)+dimy_B];
		M_d[7]=B_d[id_p+dimy_B];
		M_d[8]=B_d[(id_p+1)+dimy_B];

		for(i=0;i<msk*msk;i++)
			X+=M_d[i];
		Xprom=((float)X)/(msk*msk);

		for(i=0;i<msk*msk;i++)
			Y+=(Xprom-M_d[i])*(Xprom-M_d[i]);
		var_d[id]=Y/(msk*msk);

	}
}


//*****************funci�n main**********************

int main(int argc,char* argv[]){

	//***************declaración de variables**************

	int I,j,m,cont,tam_B, init,fin;
	init=atoi(argv[1]);
	fin=atoi(argv[2]);

	tam_B=(dimx+2)*(dimy+2);

	FILE *arch, *matrizG;

	int **A;
	int B[dimx+2][dimy+2];

	float t;
	clock_t tinicio, t_GPU;
	tinicio=clock();

	int *B_d, *B_h;
	float *var_d,*var_h;

	for(int d=init;d<=fin;d++){

		//*******************declaracion de variables***************

		B_h=(int *)malloc(sizeof(int)*tam_B);
		hipMalloc((void**)&B_d, tam_B*sizeof(int));
		var_h=(float *)malloc(sizeof(float)*tam_imag);
		hipMalloc((void**)&var_d,tam_imag*sizeof(float));

		A=(int **)malloc(sizeof(int)*dimx);
		for(i=0;i<dimx;i++)
			A[i]=(int*)malloc(sizeof(int)*dimy);

	//*****************calculo matriz B****************

	char ruta1[]="MiTesis/";
	sprintf(ruta1, "%s%d%s","RGB/",d,"/G");
	matrizG=fopen(ruta1,"r+");

	for(i=0;i<dimx;i++)
		for(j=0;j<dimy;j++)
			fscanf(matrizG, "%d", &A[i][j]);
		fclose(matrizG);

	cont=0;
	for(i=0;i<dimx+2;i++){
		for(j=0;j<dimy+2;j++){
			B[i][j]=((i==0 || j==0 || i==dimx+1 || j==dimy+1) ? 0:A[i-1][j-1]);
			B_h[cont]=B[i][j];
			cont++;

		}

	}

	//******************llamado de kernel*******************

	dim3 Grid(128,96);
	dim3 Block(20,20);

	hipMemcpy(B_d,B_h,sizeof(int)*tam_B,hipMemcpyHostToDevice);

	kernel<<<Grid,Block>>>(B_d,var_d);

	hipMemcpy(var_h,var_d,sizeof(float)*tam_imag,hipMemcpyDeviceToHost);


	//****************almacenamiento matriz de varianza**************

	char rutaV[]="VARIANZAS/";
	sprintf(rutaV, "%s%d", rutaV,d);
	arch=fopen(rutaV,"w+");

	for(m=0;m<tam_imag;m++){
		if(m%dimy==0 && m!=0){
			fprintf(arch,"\n");
		}
		fprintf(arch,"%f",var_h[m]);
	}

	fclose(arch);
	free(B_h);
	free(var_h);
	free(A);
	hipFree(var_d);
	hipFree(B_d);
}

// ???????????

t_GPU=clock();
t = ((float)t_GPU-(float)tinicio)/CLOCKS_PER_SEC;
printf("tiempo de procesamiento para calcular varianzas de %d matrices: %6.3fs\n",fin-init+1,t);

return 0;

}//FIN funci�n main()
